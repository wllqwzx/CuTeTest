#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cute/algorithm/copy.hpp>
#include <cute/algorithm/gemm.hpp>
#include <cute/container/tuple.hpp>
#include <cute/int_tuple.hpp>
#include <iostream>

#include <hip/hip_fp16.h>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>
#include <cutlass/gemm/dispatch_policy.hpp>

#define p(v) std::cout << #v << " : " << v << std::endl
#define kp(v) cute::print(#v " : "); cute::print(v); cute::print("\n")
#define pp(v) if (cute::thread0()) { printf("L%u : ", __LINE__); cute::print(#v " : "); cute::print(v); cute::print("\n"); }

// #define p(v)
// #define kp(v)
// #define pp(v)


void test_int_tuple() {
    // auto tup = cute::IntTuple<int, uint32_t, size_t>({1, 2U, 3U});
    auto tup = cute::make_tuple(1, 2U, 3U);
    p(tup);               // (1,2,3)
    p(cute::get<2>(tup)); // 3
    p(cute::rank(tup));   // _3
    p(cute::depth(tup));  // _1
    p(cute::size(tup));   // 6

    // auto tup_nest = cute::IntTuple<int, cute::IntTuple<int, int>>({2, cute::IntTuple<int, int>({3, 4})});
    auto tup_nest = cute::make_tuple(2, cute::make_tuple(3, cute::Int<4>{}));
    p(tup_nest);               // (2,(3,4))
    p(cute::get<1>(tup_nest)); // (3,4)
    p(cute::rank(tup_nest));   // _2
    p(cute::depth(tup_nest));  // _2
    p(cute::size(tup_nest));   // 24
}

void test_layout() {
    // auto ly = cute::Layout<cute::IntTuple<int, int, int>, cute::IntTuple<int, int, int>>({3,4,5}, {1,3,12});
    // auto ly = cute::make_layout(cute::make_shape(3,4,5), cute::make_stride(1,3,12));
    auto ly = cute::make_layout(cute::make_shape(3, 4, 5)); // use default stride
    p(ly);                // (3,4,5):(_1,3,12)
    p(cute::size<0>(ly)); // 3
    p(cute::size<1>(ly)); // 4
    p(cute::size<2>(ly)); // 5
    p(cute::size(ly));    // 60
    p(cute::cosize(ly));  // 60 = ly(cute::size(ly) - 1) + 1, last elem address
    p(ly.shape());        // (3,4,5)
    p(ly.stride());       // (_1,3,12)
    p(ly(0, 0, 0));       // 0
    p(ly(1, 0, 0));       // 1
    p(ly(0, 1, 0));       // 3
    p(ly(0, 0, 1));       // 12
    p(ly(2, 3, 4));       // 59
    p(ly(0));             // 0
    p(ly(1));             // 1
    p(ly(59));            // 59

    // assume ly = (a1, a2):(s1, s2), then ly(x1, x2) = x1 * s1 + x2 * s2
    // therefore, transpose only need to change the order of shape and stride, and
    // the memory can keep unchanged
    // ((a1, a2):(s1, s2))(x1, x2) = ((a2, a1):(s2, s1))(x2, x1)

    // show 2d coord to offset ascii view, only support 2d layout
    cute::print_layout(cute::make_layout(cute::make_shape(8, 4), cute::make_shape(4, 2)));
    // or print_latex for latex source
    // cute::print_latex(cute::make_layout(cute::make_shape(8, 4), cute::make_shape(4, 2)));
}

void test_layout_opeartion() {
    auto nest_ly = cute::make_layout(cute::make_shape(4, cute::make_shape(4, 2)),
                                     cute::make_shape(4, cute::make_shape(1, 16)));
    p(nest_ly);                 // (4,(4,2)):(4,(1,16))
    p(cute::flatten(nest_ly));  // (4,4,2):(4,1,16)

    // The coalesce operation first flattens the layout, then combines all the modes that are possible to
    // combine, starting with mode 0 (the leftmost mode) and moving right.
    p(cute::coalesce(nest_ly));     // (4,4,2):(4,1,16)
    auto nest_ly2 = cute::make_layout(cute::make_shape(cute::_8{}, cute::_4{}));
    p(cute::coalesce(nest_ly2));    // _32:_1

    // composition
    //   - composition(ly_a, ly_b)(idx) == ly_a(ly_b(idx))
    //   - layout maps coord (or idx) to offset
    //   - compose ly_a with ly_b requires ly_a's max input idx cover ly_b's max output offset
    //   - not any two layouts can be composed, because the result mapping may not be able to be
    //     represented with shape + stride
    using namespace cute;
    auto ly_a = make_layout(make_shape(Int<20>{}, _2{}), make_stride(_16{}, _4{}));
    auto ly_b = make_layout(make_shape(     _4{}, _5{}), make_stride( _1{}, _4{}));
    auto ly_ab = composition(ly_a, ly_b);
    p(ly_ab);           // (_4,_5):(_16,_64)
    p(ly_ab(5));        // 80
    p(ly_a(ly_b(5)));   // 80

    auto a = make_layout(Shape<_4,_3>{}, Stride<_3,_1>{});
    auto b = make_layout(Shape<_24>{});
    p(composition(a, b));

    // layout products: reproduce one layout over another
    Layout tile            = Layout<Shape <_2,_2>,
                                    Stride<_1,_2>>{};
    Layout matrix_of_tiles = Layout<Shape <_3,_4>,
                                    Stride<_4,_1>>{};

    p(tile);                                    // (_2,_2):(_1,_2)
    p(matrix_of_tiles);                         // (_3,_4):(_4,_1)
    p(logical_product(tile, matrix_of_tiles));  // ((_2,_2),(_3,_4)):((_1,_2),(_16,_4))
    p(blocked_product(tile, matrix_of_tiles));  // ((_2,_3),_8):((_1,_16),_2)
    p(raked_product(tile, matrix_of_tiles));    // ((_3,_2),(_4,_2)):((_16,_1),(_4,_2))
    p(tiled_product(tile, matrix_of_tiles));    // ((_2,_2),_3,_4):((_1,_2),_16,_4)

    // layout division: divide a layout into components, are useful as a basis for tiling and partitioning layouts.
    Layout full_layout = Layout<Shape<_16>, Stride<_3>>{};
    Layout tile_shape = Layout<Shape<_4>, Stride<_1>>{};
    p(full_layout);                                 // (_16):(_3)
    p(tile_shape);                                  // (_4):(_1)
    p(logical_divide(full_layout, tile_shape));     // ((_4),_4):((_3),_12)
    p(zipped_divide(full_layout, tile_shape));
    // p(tiled_divide(full_layout, tile_shape));
}

void test_swizzle() {
    using namespace cute;
    auto a = Layout<Shape<_8,_8>, Stride<_8,_1>>{};
    cute::print_layout(a);
    cute::print_layout(composition(Swizzle<0,0,0>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{})); // identity
    cute::print_layout(composition(Swizzle<0,0,7>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{})); // identity
    cute::print_layout(composition(Swizzle<0,1,0>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{})); // identity
    cute::print_layout(composition(Swizzle<0,7,0>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{})); // identity
    cute::print_layout(composition(Swizzle<1,0,1>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<1,0,2>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<1,0,3>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<1,1,1>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<1,2,1>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<2,0,3>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<2,0,-3>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<2,1,-3>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
    cute::print_layout(composition(Swizzle<3,3,3>{}, Layout<Shape<_8,_8>, Stride<_8,_1>>{}));
}

void __global__ test_debug_kernel() {
    if (cute::thread0()) {
        cute::print("thread=%d, block=%d\n", threadIdx.x, blockIdx.x); // thread=0, block=0
    }
    __syncthreads();
    if (cute::thread(1)) {
        cute::print("thread=%d, block=%d\n", threadIdx.x, blockIdx.x); // thread=1, block=0
    }
    __syncthreads();
    if (cute::thread(5, 1)) {
        cute::print("thread=%d, block=%d\n", threadIdx.x, blockIdx.x); // thread=5, block=1
    }
}

void test_debug() {
    test_debug_kernel<<<2, 32>>>();
    hipDeviceSynchronize();
}

void __global__ test_tensor_kernel(void* ptr) {
    // global memory
    if (cute::thread0()) {
        cute::print("global memory\n");
        // tag memory
        auto gmem_ptr = cute::make_gmem_ptr(reinterpret_cast<float*>(ptr));
        cute::print(gmem_ptr);      // gmem_ptr_32b((nil))
        cute::print("\n");

        // static layout
        cute::Tensor gmem_8s = cute::make_tensor(gmem_ptr, cute::Int<8>{});
        cute::print(gmem_8s);       // _8:_1
        cute::print("\n");

        // dynamic layout
        cute::Tensor gmem_8d = cute::make_tensor(gmem_ptr, 8);
        cute::print(gmem_8d);       // 8:_1
        cute::print("\n");

        // mixed static & dynamic
        cute::Tensor gmem_8dx16s = make_tensor(gmem_ptr, cute::make_shape (      8    , cute::_16{}),
                                                         cute::make_stride(cute::_16{}, cute::_1{}));
        cute::print(gmem_8dx16s);           // (8,_16):(_16,_1) with tensor value
        cute::print(gmem_8dx16s.layout());  // (8,_16):(_16,_1)
        cute::print("\n");
    }
    __syncthreads();

    // shared memory
    if (cute::thread0()) {
        cute::print("shared memory\n");
        auto smem_shape = cute::make_shape(cute::_4{}, cute::_8{});
        __shared__ float smem[decltype(cute::size(smem_shape))::value];   // (static-only allocation)
        cute::Tensor smem_4x8_col = cute::make_tensor(cute::make_smem_ptr(&smem[0]), smem_shape);
        cute::Tensor smem_4x8_row = cute::make_tensor(cute::make_smem_ptr(&smem[0]), smem_shape, cute::GenRowMajor{});
        cute::print(smem_4x8_col);  // (_4,_8):(_1,_4)
        cute::print("\n");
        cute::print(smem_4x8_row);  // (_4,_8):(_8,_1)
        cute::print("\n");
    }
    __syncthreads();

    // register memory (static layouts only)
    if (cute::thread0()) {
        cute::print("register memory\n");
        cute::Tensor rmem_4x8_col = cute::make_tensor<float>(cute::make_shape(cute::_4{}, cute::_8{}));
        cute::Tensor rmem_4x8_row = cute::make_tensor<float>(cute::make_shape(cute::_4{}, cute::_8{}), cute::GenRowMajor{});
        cute::Tensor rmem_4x8_mix = cute::make_tensor<float>(cute::make_shape(cute::_4{}, cute::_8{}), cute::make_stride(cute::_2{}, cute::_32{}));
        cute::print(rmem_4x8_col);  // (_4,_8):(_1,_4)
        cute::print("\n");
        cute::print(rmem_4x8_row);  // (_4,_8):(_8,_1)
        cute::print("\n");
        cute::print(rmem_4x8_mix);  // (_4,_8):(_2,_32)
        cute::print("\n");

        // slice with _ (only create a view)
        auto gmem_ptr = cute::make_gmem_ptr(reinterpret_cast<float*>(ptr));
        cute::Tensor gmem_8dx16s = make_tensor(gmem_ptr, cute::make_shape (      8    , cute::_16{}),
                                                         cute::make_stride(cute::_16{}, cute::_1{}));
        cute::Tensor gmem_16s = gmem_8dx16s(1, cute::_); // slice
        cute::print(gmem_16s);      // (_16):(_1)
        cute::print("\n");

        // make_fragment_like (static src layouts only), inherit layout and dtype
        cute::Tensor rmem_16s = cute::make_fragment_like(gmem_16s);
        cute::print(rmem_16s);      // (_16):(_1)
        cute::print("\n");
        // if we only want reuse layout but not dtype, use make_tensor:
        cute::Tensor rmem_16h = cute::make_tensor<cutlass::half_t>(make_layout_like(gmem_16s.layout()));
        cute::print(rmem_16h);      // (_16):(_1)
        cute::print("\n");
    }

    // global => register copy
    cute::Tensor gmem = cute::make_tensor(cute::make_gmem_ptr(reinterpret_cast<float*>(ptr)),
                                                              cute::make_shape(cute::_8{}, 16));
    CUTE_STATIC_ASSERT_V(cute::rank(gmem) == cute::_2{});
    CUTE_STATIC_ASSERT_V(cute::is_static<decltype(cute::shape<0>(gmem))>{});
    cute::Tensor rmem_8 = cute::make_fragment_like(gmem(cute::_, 0)); // (_8):(_1)
    for (int t = 0; t < cute::size<1>(gmem); ++t) {
        // global => regster
        cute::copy(gmem(cute::_, t), rmem_8);
        // do_something(rmem_8)
    }
}

void test_tensor() {
    // CuTe's Tensor class represents a multidimensional array. 
    // The array's elements can live in any kind of memory, 
    // including global memory, shared memory, and register memory.

    //  access a Tensor's elements
    // operator(a, b, c);
    // operator(Coord({a, b, c}));
    // operator[Coord({a, b, c})];

    // One could summarize almost all CuTe use cases as follows:
    // - create Layouts,
    // - create Tensors with those Layouts, and
    // - invoke (either CuTe's, or custom) algorithms on those Tensors.

    // Users can "tag" the memory with its space
    // e.g., global or shared by calling make_gmem_ptr(g) when g is a pointer to 
    // global memory, or make_smem_ptr(s) when s is a pointer to shared memory.

    // Tagging memory makes it possible for CuTe's Tensor algorithms to use 
    // the fastest implementation for the specific kind of memory. It also avoids 
    // incorrect memory access.

    float* ptr;
    hipMalloc(&ptr, sizeof(float) * 128);
    test_tensor_kernel<<<1,32>>>(ptr);
    hipDeviceSynchronize();
}

void test_algorithm() {
    // common numerical algorithms performed on Tensors

    // 1. copy
    // The copy algorithm has two main overloads.
    // - copy(src, dst) : use default implementation 
    // - copy(copy_atom, src, dst) : use user provided copy impl: copy_atom

    // Either the default implementation or the implementation selected by a Copy_Atom
    // overload may use none or all available parallelism, and may have a variety of
    // synchronization semantics. The behavior depends on copy's parameter types.

    // users will need to perform the additional synchronization appropriate to that underlying
    // implementation before they may use the results of the copy algorithm

    // 2. copy_if
    // 3. gemm
    // 4. axpby
    // 5. fill
    // 6. clear
}

void __global__ test_mma_atom_kernel() {
    // template <class MMA_Atom,
    //         class AtomLayoutMNK   = Layout<Shape<_1,_1,_1>>,
    //         class ValLayoutMNK    = Layout<Shape<_1,_1,_1>>,
    //         class PermutationsMNK = Tile<Underscore,Underscore,Underscore>>
    // struct TiledMMA : MMA_Atom;

    // Here, the AtomLayoutMNK is the "thread" tiling of the atom -- how many replicates of this MMA atom do you want to tile across the logical MNK modes by distinct threads.
    // The ValLayoutMNK similarly specific the tiling across replicate values instead -- how many atoms is each thread going to issue as a part of this tiled MMA.


    using namespace cute;

    constexpr int kNWarps = 2;
    using MMA_Atom_Arch = MMA_Atom<SM80_16x8x16_S32S8S8S32_TN>;
    using TiledMma = TiledMMA<
        MMA_Atom_Arch,                      // MMA_atom:                                    16x8x16
        Layout<Shape<Int<kNWarps>,_1,_1>>,  // MMA_atom * AtomLayoutMNK:                    
        Layout<Shape<_1,_2,_2>>>;           // MMA_atom * AtomLayoutMNK * ValLayoutMNK:     

    // dummy smem tensor
    __shared__ int8_t smem_buf[128*32];
    using BLK_M = _32;
    using BLK_N = _16;
    using BLK_K = _32;

    Tensor sA = make_tensor(make_smem_ptr(smem_buf), Layout<Shape<BLK_M, BLK_K>>{}); // (BLK_M, BLK_K)
    Tensor sB = make_tensor(make_smem_ptr(smem_buf), Layout<Shape<BLK_N, BLK_K>>{}); // (BLK_N, BLK_K)
    pp(sA.layout());
    pp(sB.layout());

    const int thread_idx = 0;
    TiledMma tiled_mma;
    auto thr_mma = tiled_mma.get_thread_slice(thread_idx);
    Tensor tCrA = thr_mma.partition_fragment_A(sA);                     // (MMA, N_MMA_M, N_MMA_K)
    Tensor tCrB = thr_mma.partition_fragment_B(sB);                     // (MMA, N_MMA_N, N_MMA_K)
    pp(tCrA.layout());
    pp(tCrB.layout());
}


void test_mma_atom() {
    // MMAs are architecture-specific. Different generations of GPU architectures introduce different sets of 
    // MMA instructions. However, CuTe features such as Layout makes it possible to expose MMAs for use in generic 
    // CUDA C++ code. We do this in two steps:

    // - We wrap each MMA's PTX instruction in an "Operation" struct.
    // - For each Operation struct, we define a "Traits" struct that defines all of the meta-information needed to use 
    //   the Operation.

    // CuTe supports MMA atoms that operate at a variety of hardware levels, including
    // - a single thread (e.g., fused multiply-add (FMA) instruction);
    // - a quadpair (Volta);
    // - a single warp (Ampere); and
    // - a warpgroup (Hopper).

    // example:
    test_mma_atom_kernel<<<1, 32>>>();
    hipDeviceSynchronize();
}

template<class Mshape, class NShape, class KShape,
         class TA, class AStride, class ABlockLayout, class AThreadLayout,
         class TB, class BStride, class BBlockLayout, class BThreadLayout,
         class TC, class CStride, class CBlockLayout, class CThreadLayout>
__global__ void __launch_bounds__(decltype(cute::size(CThreadLayout{}))::value)
gemm_kernel(const Mshape M, const NShape N, const KShape K,
            const TA* __restrict__ A, const AStride dA, const ABlockLayout sA, const AThreadLayout tA,
            const TB* __restrict__ B, const BStride dB, const BBlockLayout sB, const BThreadLayout tB,
                  TC* __restrict__ C, const CStride dC, const CBlockLayout sC, const CThreadLayout tC) {
    // check
    CUTE_STATIC_ASSERT(cute::is_static<ABlockLayout>::value);
    CUTE_STATIC_ASSERT(cute::is_static<BBlockLayout>::value);
    CUTE_STATIC_ASSERT(cute::is_static<CBlockLayout>::value);
    CUTE_STATIC_ASSERT(cute::is_static<AThreadLayout>::value);
    CUTE_STATIC_ASSERT(cute::is_static<BThreadLayout>::value);
    CUTE_STATIC_ASSERT(cute::is_static<CThreadLayout>::value);

    CUTE_STATIC_ASSERT(cute::size(tA) == cute::size(tC));
    CUTE_STATIC_ASSERT(cute::size(tB) == cute::size(tC));

    CUTE_STATIC_ASSERT(cute::shape<0>(sA) == cute::shape<0>(sC));   // BLK_M
    CUTE_STATIC_ASSERT(cute::shape<0>(sB) == cute::shape<1>(sC));   // BLK_N
    CUTE_STATIC_ASSERT(cute::shape<1>(sA) == cute::shape<1>(sB));   // BLK_K

    if (cute::thread0()) {
        kp(sA); // (_128,_8):(_1,_128)
        kp(sB); // (_128,_8):(_1,_128)
        kp(sC); // (_128,_128):(_1,_128)
        kp(tA); // (_32,_8):(_1,_32)
        kp(tB); // (_32,_8):(_1,_32)
        kp(tC); // (_16,_16):(_1,_16)
    }

    // alloc shared memory buffers
    __shared__ TA smemA_ptr[decltype(cute::cosize(sA))::value]; // BLK_M * BLK_K
    __shared__ TB smemB_ptr[decltype(cute::cosize(sB))::value]; // BLK_N * BLK_K

    auto smemA = cute::make_tensor(cute::make_smem_ptr(smemA_ptr), sA);
    auto smemB = cute::make_tensor(cute::make_smem_ptr(smemB_ptr), sB);

    // Represent the full tensors
    auto gmemA = cute::make_tensor(cute::make_gmem_ptr(A), cute::make_layout(cute::make_shape(M, K)));
    auto gmemB = cute::make_tensor(cute::make_gmem_ptr(B), cute::make_layout(cute::make_shape(N, K)));
    auto gmemC = cute::make_tensor(cute::make_gmem_ptr(C), cute::make_layout(cute::make_shape(M, N)));

    if (cute::thread0()) {
        kp(smemA.layout()); // (_128,_8):(_1,_128)
        kp(smemB.layout()); // (_128,_8):(_1,_128)

        kp(gmemA.layout()); // (1024,1024):(_1,1024)
        kp(gmemB.layout()); // (1024,1024):(_1,1024)
        kp(gmemC.layout()); // (1024,1024):(_1,1024)
    }

    //
    // Get the corresponding tiles for this thread block
    //

    // auto blk_shape = cute::make_shape(cute::size<0>(smemA), cute::size<0>(smemB), cute::size<1>(smemB));    // (BLK_M, BLM_N, BLK_K)
    // auto blk_coord = cute::make_coord(blockIdx.x, blockIdx.y, cute::_);

    // (M, K) => (BLK_M, BLK_K, ceil_div(K, BLK_K))
    // cute::Tensor gA = cute::local_tile(gmemA, blk_shape, blk_coord, cute::Step<cute::_1, cute::Underscore, cute::_1>());
    cute::Tensor gA = cute::local_tile(gmemA, smemA.layout().shape(), cute::make_coord(blockIdx.x, cute::_));

    // (N, K) => (BLK_N, BLK_K, ceil_div(K, BLK_K))
    // cute::Tensor gB = cute::local_tile(gmemB, blk_shape, blk_coord, cute::Step<cute::Underscore, cute::_1, cute::_1>());
    cute::Tensor gB = cute::local_tile(gmemB, smemB.layout().shape(), cute::make_coord(blockIdx.y, cute::_));

    // (M, N) => (BLK_M, BLK_N)
    // cute::Tensor gC = cute::local_tile(gmemC, blk_shape, blk_coord, cute::Step<cute::_1, cute::_1, cute::Underscore>());
    cute::Tensor gC = cute::local_tile(gmemC, 
                               cute::make_shape(cute::size<0>(smemA), cute::size<0>(smemB)),
                               cute::make_coord(blockIdx.x, blockIdx.y));


    if (cute::thread0()) {
        kp(gA.layout());    // (_128,_8,128):(_1,1024,8192)
        kp(gB.layout());    // (_128,_8,128):(_1,1024,8192)
        kp(gC.layout());    // (_128,_128):(_1,1024)
    }

    // 
    // thread binding for copying: 
    //      partition A,B gmem and smem tile for each thread in thread layout A and B
    // 

    // global tile: (BLK_M, BLK_K, ceil_div(K, BLK_K)) => (THR_M, THR_K, ceil_div(K, BLK_K))
    //              (128, 8, 128) / (32, 8) => (4, 1, 128)
    cute::Tensor tAgA = cute::local_partition(gA, tA, threadIdx.x);
    // shm tile: (BLK_M, BLK_K) => (THR_M, THR_K)
    //           (128, 8) / (32, 8) => (4, 1)
    cute::Tensor tAsA = cute::local_partition(smemA, tA, threadIdx.x);

    // ditto for B
    cute::Tensor tBgB = cute::local_partition(gB, tB, threadIdx.x);
    cute::Tensor tBsB = cute::local_partition(smemB, tB, threadIdx.x);

    if (cute::thread0()) {
        kp(tAgA.layout());  // (_4,_1,128):(_32,_0,8192)
        kp(tAsA.layout());  // (_4,_1):(_32,_0)
        kp(tBgB.layout());  // (_4,_1,128):(_32,_0,8192)
        kp(tBsB.layout());  // (_4,_1):(_32,_0)
    }

    // 
    // thread binding for computing:
    //      partition A,B,C smem tile for each thread in thread layout C
    // 

    // Partition smemA (BLK_M, BLK_K) by the rows of tC:
    // - (BLK_M, BLK_K) => (THR_M, BLK_K)
    // - (128, 8) / (16, _) => (8, 8)
    cute::Tensor tCsA = cute::local_partition(smemA, tC, threadIdx.x, cute::Step<cute::_1, cute::Underscore>{});

    // Partition smemA (BLK_N, BLK_K) by the cols of tC:
    // - (BLK_N, BLK_K) => (THR_N,BLK_K)
    // - (128, 8) / (_, 16) => (8, 8)
    cute::Tensor tCsB = cute::local_partition(smemB, tC, threadIdx.x, cute::Step<cute::Underscore, cute::_1>{});

    // Partition gmemC (BLK_M, BLK_N) by the tile of tC:
    // - (BLK_M, BLK_N) => (THR_M, THR_N)
    // - (128, 128) / (16, 16) => (8, 8)
    // This is an exception, tCgC is for copying from regC to globalC
    cute::Tensor tCgC = cute::local_partition(gC, tC, threadIdx.x, cute::Step<cute::_1, cute::_1>{});

    // alloc register C for each thread for computing
    cute::Tensor tCrC = cute::make_fragment_like(tCgC);
    // set the accumulators to 0
    cute::clear(tCrC);

    if (cute::thread0()) {
        kp(tCsA.layout());  // (_8,_8):(_16,_128)
        kp(tCsB.layout());  // (_8,_8):(_16,_128)
        kp(tCgC.layout());  // (_8,_8):(_16,16384)
        kp(tCrC.layout());  // (_8,_8):(_1,_8)
    }

    // 
    // GEMM main loop
    // 

    auto k_iter = cute::size<2>(tAgA); // ceil_div(K, BLK_K)
    auto _ = cute::Underscore{};
    for (int k=0; k<k_iter; ++k) {
        // Copy gmem to smem
        cute::copy(tAgA(_,_,k), tAsA);
        cute::copy(tBgB(_,_,k), tBsB);

        // In case copy uses cp.async
        cute::cp_async_fence();     // cp.async.commit_group
        cute::cp_async_wait<0>();   // cp.async_wait_group 0

        __syncthreads();

        // Compute gemm on smem A & B and reg C
        cute::gemm(tCsA, tCsB, tCrC);
    }

    // epilogue: tCgC = alpha * tCrC + beta * tCgC => tCgC = tCrC
    cute::axpby((TC)1.0, tCrC, (TC)0.0, tCgC);

    // or just copy tCrC back to tCgC directly
    // cute::copy(tCrC, tCgC);
}

template<typename TA, typename TB, typename TC>
void gemm(TA* a_ptr, TB* b_ptr, TC* c_ptr, const int M, const int N, const int K) {
    // device tensor layout
    // - A: MxK (M-major)
    // - B: NxK (N-major)
    // - C: M*N (M-major)

    // Define global strides (mixed)
    auto dA = cute::make_stride(cute::_1{}, M);
    auto dB = cute::make_stride(cute::_1{}, N);
    auto dC = cute::make_stride(cute::_1{}, M);

    // Define block sizes (static)
    auto bM = cute::_128{};
    auto bN = cute::_128{};
    auto bK = cute::_8{};

    // Define the block layouts (static), use default stride
    auto sA = cute::make_layout(cute::make_shape(bM, bK));
    auto sB = cute::make_layout(cute::make_shape(bN, bK));
    auto sC = cute::make_layout(cute::make_shape(bM, bN));

    // Define the thread layouts (static), use 8 warp
    auto tA = cute::make_layout(cute::make_shape(cute::_32{}, cute::_8{}));
    auto tB = cute::make_layout(cute::make_shape(cute::_32{}, cute::_8{}));
    auto tC = cute::make_layout(cute::make_shape(cute::_16{}, cute::_16{}));

    dim3 dimGrid(cute::ceil_div(M, bM), cute::ceil_div(N, bN));
    dim3 dimBlock(cute::size(tC)); // 32 * 8 = 256

    hipStream_t stream = 0;
    gemm_kernel<<<dimGrid, dimBlock, 0, stream>>>(
        M, N, K,
        a_ptr, dA, sA, tA,
        b_ptr, dB, sB, tB,
        c_ptr, dC, sC, tC
    );
}

void test_gemm_kernel() {
    constexpr int M = 1024;
    constexpr int N = 1024;
    constexpr int K = 1024;

    using TA = cutlass::half_t;
    using TB = cutlass::half_t;
    using TC = cutlass::half_t;

    size_t size_a = M * K * sizeof(TA);
    size_t size_b = K * N * sizeof(TB);
    size_t size_c = M * N * sizeof(TC);

    TA* h_a = (TA*) malloc(size_a);
    TB* h_b = (TB*) malloc(size_b);
    TC* h_c = (TC*) malloc(size_c);
    TC* h_c_ref = (TC*) malloc(size_c);

    TA* d_a;
    TB* d_b;
    TC* d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    srand(time(0));
    for (int i=0; i<M*K; ++i) {
        h_a[i] = (TA)(rand() / float(RAND_MAX));
    }
    for (int i=0; i<K*N; ++i) {
        h_b[i] = (TB)(rand() / float(RAND_MAX));
    }
    for (int i=0; i<M*N; ++i) {
        h_c[i] = (TC)(0);
    }
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gemm(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    CUTE_CHECK_LAST();

    // check error
    // for (int im=0; im<M; ++im) {
    //     for (int in=0; in<N; ++in) {
    //         TC acc = (TC)0.0;
    //         for (int ik=0; ik<K; ++ik) {
    //             acc += h_a[im + ik * M] * h_b[in + ik * K];
    //         }
    //         double rel_err = cute::abs((double)h_c[im + in * M] - (double)acc) / (double)acc;
    //         assert (rel_err < 0.03);
    //     }
    // }

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_ref);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

template<typename TileShape, class TA, class TB, class TC,
        class TiledMma,
        class GmemTiledCopyA, class SmemLayoutAtomA, class SmemCopyAtomA,
        class GmemTiledCopyB, class SmemLayoutAtomB, class SmemCopyAtomB>
__global__ void gemm_kernel_with_atom(int M, int N, int K,
            const TA* __restrict__ A, const TB* __restrict__ B, TC* __restrict__ C) {
    using namespace cute;

    pp(TileShape{});        // (_128,_128,_64)
    pp(SmemLayoutAtomA{});  // S<2,4,3> o _0 o (_16,_64):(_64,_1)
    pp(SmemLayoutAtomB{});  // S<2,4,3> o _0 o (_16,_64):(_64,_1)

    // =====================
    // multi stage example
    // using DispatchPolicy = cutlass::gemm::MainloopSm80CpAsync<3>;
    // using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtomA{},
    //                                            make_shape(shape<0>(TileShape{}), shape<2>(TileShape{}),
    //                                            Int<DispatchPolicy::Stages>{})));
    // pp(SmemLayoutA{});  // S<2,4,3> o _0 o (_128,_64,_3):(_64,_1,_8192)

    using SmemLayoutA = decltype(tile_to_shape(
        SmemLayoutAtomA{},
        make_shape(shape<0>(TileShape{}), shape<2>(TileShape{}))));
    using SmemLayoutB = decltype(tile_to_shape(
        SmemLayoutAtomB{},
        make_shape(shape<1>(TileShape{}), shape<2>(TileShape{}))));

    pp(SmemLayoutA{});  // S<2,4,3> o _0 o (_128,_64):(_64,_1) : (BLK_M, BLK_K)
    pp(SmemLayoutB{});  // S<2,4,3> o _0 o (_128,_64):(_64,_1) : (BLK_N, BLK_K)

    static_assert(rank(SmemLayoutA{}) == 2);
    static_assert(rank(SmemLayoutB{}) == 2);

    // alloc shared memory buffers
    __shared__ TA smemA_ptr[decltype(cute::cosize(SmemLayoutA{}))::value]; // BLK_M * BLK_K
    __shared__ TB smemB_ptr[decltype(cute::cosize(SmemLayoutB{}))::value]; // BLK_N * BLK_K

    Tensor sA = make_tensor(make_smem_ptr(smemA_ptr), SmemLayoutA{});
    Tensor sB = make_tensor(make_smem_ptr(smemB_ptr), SmemLayoutB{});
    pp(sA.layout());    // S<2,4,3> o _0 o (_128,_64):(_64,_1) : (BLK_M, BLK_K)
    pp(sB.layout());    // S<2,4,3> o _0 o (_128,_64):(_64,_1) : (BLK_N, BLK_K)

    // Represent the full tensors
    auto gmemA = cute::make_tensor(cute::make_gmem_ptr(A), cute::make_layout(cute::make_shape(M, K), cute::GenRowMajor{}));
    auto gmemB = cute::make_tensor(cute::make_gmem_ptr(B), cute::make_layout(cute::make_shape(N, K), cute::GenRowMajor{}));
    auto gmemC = cute::make_tensor(cute::make_gmem_ptr(C), cute::make_layout(cute::make_shape(M, N), cute::GenRowMajor{}));

    pp(gmemA.layout()); // (1024,1024):(1024,_1)
    pp(gmemB.layout()); // (1024,1024):(1024,_1)
    pp(gmemC.layout()); // (1024,1024):(1024,_1)

    // Get the corresponding tiles for this thread block
    // (M, K) => (BLK_M, BLK_K, ceil_div(K, BLK_K))
    cute::Tensor gA = cute::local_tile(gmemA, sA.layout().shape(), cute::make_coord(blockIdx.x, cute::_));

    // (N, K) => (BLK_N, BLK_K, ceil_div(K, BLK_K))
    cute::Tensor gB = cute::local_tile(gmemB, sB.layout().shape(), cute::make_coord(blockIdx.y, cute::_));

    // (M, N) => (BLK_M, BLK_N)
    cute::Tensor gC = cute::local_tile(gmemC, 
                                       cute::make_shape(cute::size<0>(sA), cute::size<0>(sB)),
                                       cute::make_coord(blockIdx.x, blockIdx.y));

    pp(gA.layout());    // (_128,_64,16):(1024,_1,_64)
    pp(gB.layout());    // (_128,_64,16):(1024,_1,_64)
    pp(gC.layout());    // (_128,_128):(1024,_1)


    // threading binding for gmem=>smem copy using GmemTiledCopyA/B
    const int thread_idx = threadIdx.x;
    GmemTiledCopyA gmem_tiled_copy_a;
    GmemTiledCopyB gmem_tiled_copy_b;
    auto copy_a_thr = gmem_tiled_copy_a.get_slice(thread_idx);
    auto copy_b_thr = gmem_tiled_copy_b.get_slice(thread_idx);

    Tensor tAgA = copy_a_thr.partition_S(gA);
    Tensor tAsA = copy_a_thr.partition_D(sA);
    Tensor tBgB = copy_b_thr.partition_S(gB);
    Tensor tBsB = copy_b_thr.partition_D(sB);
    pp(tAsA.layout());  // ((_16,_1),_4,_1):((_1,_0),_2048,_0)         : (ACPY, ACPY_M, ACPY_K)
    pp(tBsB.layout());  // ((_16,_1),_4,_1):((_1,_0),_2048,_0)         : (BCPY, BCPY_N, BCPY_K)
    pp(tAgA.layout());  // ((_16,_1),_4,_1,16):((_1,_0),32768,_0,_64)  : (ACPY, ACPY_M, ACPY_K, k_loop)
    pp(tBgB.layout());  // ((_16,_1),_4,_1,16):((_1,_0),32768,_0,_64)  : (BCPY, BCPY_N, BCPY_K, k_loop)

    // 
    // MMA compute
    // 
    TiledMma tiled_mma;
    auto thr_mma = tiled_mma.get_thread_slice(thread_idx);
    // allocate regA and regB for each thread
    Tensor tCrA  = thr_mma.partition_fragment_A(sA);
    Tensor tCrB  = thr_mma.partition_fragment_B(sB);
    pp(tCrA.layout());  // ((_4,_2,_2),_4,_2):((_1,_4,_8),_16,_64)  : (MMA_elem_per_thr, BLK_M/WP_M/MMA_M, BLK_K/WP_K/MMA_K)
    pp(tCrB.layout());  // ((_4,_2),_8,_2):((_1,_4),_8,_64)         : (MMA_elem_per_thr, BLK_N/WP_N/MMA_N, BLK_K/WP_K/MMA_K)

    // thread binding for gC
    Tensor tCgC = thr_mma.partition_C(gC);
    // allocate regC for each thread
    Tensor tCrC = thr_mma.partition_fragment_C(gC);
    pp(tCgC.layout());  // ((_2,_2),_4,_8):((_1,8192),32768,_16)
    pp(tCrC.layout());  // ((_2,_2),_4,_8):((_1,_2),_4,_16)
    cute::clear(tCrC);

    // threading binding for smem
    auto smem_tiled_copy_a = cute::make_tiled_copy_A(SmemCopyAtomA{}, tiled_mma);
    auto thr_copy_A        = smem_tiled_copy_a.get_thread_slice(thread_idx);
    Tensor tCsA            = thr_copy_A.partition_S(sA);
    Tensor tCrA_copy_view  = thr_copy_A.retile_D(tCrA);  // tCrA and tCrA_copy_view share same storage, 
                                                         // tCrA_copy_view is used for efficient smem=>rmem copy
    CUTE_STATIC_ASSERT_V(size<1>(tCsA) == size<1>(tCrA_copy_view));
    pp(tCsA.layout());              // ((_16,_1),_4,_2):((_1,_0),_2048,32)  : (MMA_elem_per_thr, BLK_M/WP_M/MMA_M, BLK_K/WP_K/MMA_K)
    pp(tCrA_copy_view.layout());    // ((_16,_1),_4,_2):((_1,_0),_16,_64)   : (MMA_elem_per_thr, BLK_M/WP_M/MMA_M, BLK_K/WP_K/MMA_K)
    // tCrA and tCrA_copy_view share same storage
    // pp(tCrA(0));            // 0
    // pp(tCrA_copy_view(0));  // 0
    // tCrA(0) = 123;
    // pp(tCrA_copy_view(0));  // 123

    auto smem_tiled_copy_b = cute::make_tiled_copy_B(SmemCopyAtomB{}, tiled_mma);
    auto thr_copy_B        = smem_tiled_copy_b.get_thread_slice(thread_idx);
    Tensor tCsB            = thr_copy_B.partition_S(sB);
    Tensor tCrB_copy_view  = thr_copy_B.retile_D(tCrB);  // tCrB and tCrB_copy_view share same storage
                                                         // tCrB_copy_view is used for efficient smem=>rmem copy
    CUTE_STATIC_ASSERT_V(size<1>(tCsB) == size<1>(tCrB_copy_view));
    pp(tCsB.layout());              // ((_16,_1),_4,_2):((_1,_0),_2048,32)  : (MMA_elem_per_thr, BLK_N/WP_N/MMA_N, BLK_K/WP_K/MMA_K)
    pp(tCrB_copy_view.layout());    // ((_16,_1),_4,_2):((_1,_0),_16,_64)   : (MMA_elem_per_thr, BLK_N/WP_N/MMA_N, BLK_K/WP_K/MMA_K)

    // 
    // GEMM main loop (outer k)
    // 
    auto k_loop = size<3>(tAgA); // ceil_div(K, BLK_K)
    for (int k=0; k<k_loop; ++k) {
        // copy gmem => smem (use LDGSTS(cp.async))
        copy(gmem_tiled_copy_a, tAgA(_,_,_,k), tAsA);
        cute::cp_async_fence();     // cp.async.commit_group
        copy(gmem_tiled_copy_b, tBgB(_,_,_,k), tBsB);
        cute::cp_async_fence();     // cp.async.commit_group
        cute::cp_async_wait<0>();   // cp.async_wait_group 0
        __syncthreads();

        // copy smem => rmem (use LDSM)
        copy(smem_tiled_copy_a, tCsA, tCrA_copy_view);
        copy(smem_tiled_copy_b, tCsB, tCrB_copy_view);

        __syncthreads();

        // tensor core mma
        cute::gemm(tiled_mma, tCrC, tCrA, tCrB, tCrC);

        // or use explicit GEMM inner loop (inner k)
        // auto k_inner_loop = size<2>(tCrA);  // BLK_K/WP_K/MMA_K
        // for (int inn_k=0; inn_k<k_inner_loop; ++inn_k) {
        //     // tensor core mma
        //     cute::gemm(tiled_mma, tCrC, tCrA(_,_,inn_k), tCrB(_,_,inn_k), tCrC);
        // }
    }

    // copy rmem=>gmem
    copy(tCrC, tCgC);
}

template<typename TA, typename TB, typename TC>
void gemm_with_atom(TA* a_ptr, TB* b_ptr, TC* c_ptr, const int M, const int N, const int K) {
    using namespace cute;
    // device tensor layout
    // - A: MxK (K-major)
    // - B: NxK (K-major)
    // - C: M*N (N-major)

    // Define block sizes (static)
    using bM = cute::_128;
    using bN = cute::_128;
    using bK = cute::_64;

    using TileShape = Shape<bM, bN, bK>;
    static constexpr int ThreadCount = 128;

    using TiledMma = TiledMMA<
        MMA_Atom<SM80_16x8x32_S32S8S8S32_TN>,
        Layout<Shape<_2,_2,_1>>,   // 2x2x1 thread group (equals to #warp)
        Layout<Shape<_1,_2,_1>>>;  // 1x2x1 value group for 16x16x32 and LDSM (seems equals to 16x16x256bit?)

    // ===== for A (M,K)  K-major =====
    using SmemLayoutAtomA = decltype(
        composition(
        Swizzle<2,4,3>{},
        Layout<Shape <_16,_64>,
               Stride<_64, _1>>{}));

    static_assert(rank(SmemLayoutAtomA{}) == 2, "SmemLayoutAtom must be rank 2 (M/N, K)");
    static_assert((size<0>(TileShape{}) % size<0>(SmemLayoutAtomA{})) == 0, "SmemLayoutAtom must evenly divide tile shape.");
    static_assert((size<2>(TileShape{}) % size<1>(SmemLayoutAtomA{})) == 0, "SmemLayoutAtom must evenly divide tile shape.");

    static constexpr int kAlignmentA = 16;
    // for gmemA=>smemA
    using GmemTiledCopyA = decltype(
        make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, int8_t>{},
                        Layout<Shape<_32,_4>, Stride<_4,_1>>{},     // thread layout, s.t. BLK_K % kAlignmentA * THR_K == 0
                        Layout<Shape<_1, Int<kAlignmentA>>>{}));    // value layout, s.t. kAlignmentA = sizeof(uint128_t) / sizeof(int8_t)

    // LDS.32- or LDSM-based copy atom
    // using SmemCopyAtomA = Copy_Atom<DefaultCopy, uint8_t>;
    // for smemA=>rmemA
    using SmemCopyAtomA = Copy_Atom<SM75_U32x4_LDSM_N, uint8_t>;   // LDSM works

    // ===== for B (N,K)  K-major =====
    using SmemLayoutAtomB = decltype(
    composition(
        Swizzle<2,4,3>{},
        Layout<Shape <_16,_64>,
                Stride<_64, _1>>{}));

    static_assert(rank(SmemLayoutAtomB{}) == 2, "SmemLayoutAtom must be rank 2 (M/N, K)");
    static_assert((size<1>(TileShape{}) % size<0>(SmemLayoutAtomB{})) == 0, "SmemLayoutAtom must evenly divide tile shape.");
    static_assert((size<2>(TileShape{}) % size<1>(SmemLayoutAtomB{})) == 0, "SmemLayoutAtom must evenly divide tile shape.");

    static constexpr int kAlignmentB = 16;
    using GmemTiledCopyB = decltype(
    make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, int8_t>{},
                    Layout<Shape<_32,_4>, Stride< _4,_1>>{},    // thread layout, s.t. BLK_K % kAlignmentB * THR_K == 0
                    Layout<Shape<_1,Int<kAlignmentB>>>{}));     // value layout, s.t. kAlignmentB = sizeof(uint128_t) / sizeof(int8_t)

    // LDS.32- or LDSM-based copy atom
    // using SmemCopyAtomB = Copy_Atom<DefaultCopy, uint32_t>;
    using SmemCopyAtomB = Copy_Atom<SM75_U32x4_LDSM_N, uint8_t>;  // LDSM works

    dim3 dimGrid(cute::ceil_div(M, bM{}), cute::ceil_div(N, bN{}));
    dim3 dimBlock(ThreadCount);

    hipStream_t stream = 0;
    gemm_kernel_with_atom<
        TileShape, TA, TB, TC,
        TiledMma,
        GmemTiledCopyA, SmemLayoutAtomA, SmemCopyAtomA,
        GmemTiledCopyB, SmemLayoutAtomB, SmemCopyAtomB
    ><<<dimGrid, dimBlock, 0, stream>>>(
        M, N, K,
        a_ptr, b_ptr, c_ptr
    );
}

void test_gemm_with_atom_kernel() {
    constexpr int M = 1024;
    constexpr int N = 1024;
    constexpr int K = 1024;

    using TA = int8_t;
    using TB = int8_t;
    using TC = int32_t;

    size_t size_a = M * K * sizeof(TA);
    size_t size_b = K * N * sizeof(TB);
    size_t size_c = M * N * sizeof(TC);

    TA* h_a = (TA*) malloc(size_a);
    TB* h_b = (TB*) malloc(size_b);
    TC* h_c = (TC*) malloc(size_c);
    TC* h_c_ref = (TC*) malloc(size_c);

    TA* d_a;
    TB* d_b;
    TC* d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    srand(time(0));
    for (int i=0; i<M*K; ++i) {
        h_a[i] = (TA)(rand() / 256 - 128);
    }
    for (int i=0; i<K*N; ++i) {
        h_b[i] = (TB)(rand() / 256 - 128);
    }
    for (int i=0; i<M*N; ++i) {
        h_c[i] = (TC)(0);
    }
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gemm_with_atom(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    CUTE_CHECK_LAST();

    // check error
    // for (int im=0; im<M; ++im) {
    //     for (int in=0; in<N; ++in) {
    //         TC acc = (TC)0.0;
    //         for (int ik=0; ik<K; ++ik) {
    //             acc += h_a[im * K + ik] * h_b[in * K + ik];
    //         }
    //         double rel_err = cute::abs((double)h_c[im * N + in] - (double)acc) / (double)acc;
    //         if ((im * N + in) % (M * N / 10) == 0) {
    //             printf("gt=%d, out=%d, at (%d,%d)\n", acc, h_c[im * N + in], im, in);
    //         }
    //         if (rel_err > 0.01) {
    //             printf("error occured: with gt=%d, out=%d, at (%d,%d)\n", acc, h_c[im * N + in], im, in);
    //             exit(1);
    //         }
    //     }
    // }

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_ref);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

template<class BlockLayout, class ThreadLayout>
__global__ void __launch_bounds__(decltype(cute::size(ThreadLayout{}))::value)
predication_kernel(const BlockLayout sA, const ThreadLayout tA) {
    // 1. make_identity_tensor:
    // - make_identity_tensor returns a "fake" tensor that when query with coord (m, n) it returns (m, n)
    cute::Tensor cA = cute::make_identity_tensor(cute::make_shape(cute::size<0>(sA), cute::size<1>(sA)));
    if (cute::thread0()) {
        kp(cA.layout());    // (_128,_8):(0:_1,1:_1)
        kp(cA(0,1));        // (0,1)
        kp(cA(2,6));        // (2,6)
        kp(cA(125,0));      // (125,0)
        kp(cA(12,7));       // (12,7)
    }
    // - this coord tensor can keep its original coord after tiling, thus can be used to trace the
    //   original coord of a tensor.
    cute::Tensor tAcA = cute::local_partition(cA, tA, threadIdx.x);
    if (cute::thread0()) {
        kp(tAcA.layout());  // (_4,_1):(0:_32,_0)
        kp(tAcA(0, 0));     // (0,0)
        kp(tAcA(1, 0));     // (32,0)
        kp(tAcA(3, 0));     // (96,0)
    }

    // - and we can use it to create a predicate tensor by comparing the coord tensor's value with
    //   the bounds of the original layout. for example:

    // predicate tensor:
    // cuye::Tensor tApA = cute::make_tensor<bool>(cute::make_shape(...), cute::make_stride(...);

    // Populate:
    // CUTE_UNROLL
    // for (int m = 0; m < cute::size<0>(tApA); ++m) {
    //     tApA(m,0) = cute::get<0>(tAcA(m,0)) < m_max_coord;
    // }

    // - We can then use the predicate tensors in copy_if to copy only the elements for which the
    //   corresponding predicate tensor elements are nonzero:

    // copy_if(tApA, tAgA(...), tAsA(...));

}

void test_predication() {
    // predication is used when tiling isn't perfect
    // The general procedure is that we:
    // - create an "identity" layout with the same shape as our original data;
    // - repeat the same tiling/partitioning/slicing (possibly rounding up) on that identity layout;
    // - create a "predicate tensor" by comparing the coordinates of that reference layout with the bounds of the original layout;
    // - use the predicate tensor to mask off accesses to out-of-bounds elements.

    auto bM = cute::_128{};
    auto bK = cute::_8{};
    auto sA = cute::make_layout(cute::make_shape(bM, bK));
    auto tA = cute::make_layout(cute::make_shape(cute::_32{}, cute::_8{}));
    dim3 dimBlock(cute::size(tA)); // 32 * 8 = 256
    predication_kernel<<<1, dimBlock>>>(sA, tA);
    hipDeviceSynchronize();
    CUTE_CHECK_LAST();
}

int main() {
    test_int_tuple();
    test_layout();
    test_layout_opeartion();
    test_swizzle();
    test_debug();
    test_tensor();
    test_algorithm();
    test_mma_atom();
    test_gemm_kernel();
    test_gemm_with_atom_kernel();
    test_predication();
}
